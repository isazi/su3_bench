#include "hip/hip_runtime.h"
#ifdef kernel_tuner
#include <lattice.hpp>
#endif

//*******************  m_mat_nn.c  (in su3.a) ****************************
//  void mult_su3_nn( su3_matrix *a,*b,*c )
//  matrix multiply, no adjoints
//  C  <-  A*B
__global__ void k_mat_nn(
    kmm::Range<int64_t> range,
    kmm::GPUSubview<site> a,
    kmm::GPUSubview<su3_matrix> b,
    kmm::GPUSubviewMut<site> c,
    int total_sites)
{
    int myThread = blockDim.x * blockIdx.x + threadIdx.x + range.begin;
    int mySite = myThread/36;

    if (mySite < range.end && mySite < total_sites) {
        int j = (myThread%36)/9;
        int k = (myThread%9)/3;
        int l = myThread%3;
        Complx cc = {0.0, 0.0};
        for (int m=0;m<3;m++)
#ifdef MILC_COMPLEX
            CMULSUM(a[mySite].link[j].e[k][m], b[j].e[m][l], cc);
#else
                cc += a[mySite].link[j].e[k][m] * b[j].e[m][l];
#endif
        c[mySite].link[j].e[k][l] = cc;
    }
}